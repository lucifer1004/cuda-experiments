#include <cstdio>
#include <cstdlib>
#include <vector>

#include <common.h>
#include <mpi.h>
#include <nvToolsExt.h>

#define N (1 << 30)

int main() {
    int rank, ranks;
    MPI_Init(nullptr, nullptr);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &ranks);
    hipSetDevice(rank);

    float *hA, *dA;

    std::string tag = "Rank " + std::to_string(rank);
    nvtxRangePush(tag.c_str());
    hipHostAlloc(&hA, sizeof(float) * N, hipHostMallocDefault);
    hipMalloc(&dA, sizeof(float) * N);
    hipMemcpy(dA, hA, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(hA, dA, sizeof(float) * N, hipMemcpyDeviceToHost);
    hipFree(dA);
    hipHostFree(hA);
    nvtxRangePop();
    MPI_Finalize();

    return 0;
}